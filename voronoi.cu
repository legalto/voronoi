#include "hip/hip_runtime.h"
#include <stdio.h>
#include "voronoi.h"
#include <vector>

__global__ void voronoi_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, COORDS_T* sites){

	//Get index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	float distance = 0.0;
	auto closest_site = 0;
	auto closest = 1.0e30f;

	// Make sure threads don't access memory outside of the image
	if((xIndex<width) && (yIndex<height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);

		#pragma unroll
		for (int index = 0; index < sites->next; ++index) {

	    		 distance = sqrtf( (xIndex - sites->x[index])*(xIndex - sites->x[index]) +
	    						  (yIndex - sites->y[index])*(yIndex - sites->y[index]) );

	    		 if (distance < closest) {
	    			closest_site = index;
	    			closest = distance;
	    		}
	    }

		// Determine pixel index of closest site
		const int site_result = sites->y[closest_site]*colorWidthStep + (3*sites->x[closest_site]);

		// Copy site pixel value into output pixel
		output[color_tid]		= input[site_result];
		output[color_tid + 1]	= input[site_result + 1];
		output[color_tid + 2]	= input[site_result + 2];

	}
}

__global__ void voronoi_kernel_shared_mem(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, COORDS_T* sites){

	__shared__ COORDS_T *sites_shared;
	sites_shared = sites;
	__syncthreads();


	//Get index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	float distance = 0.0;
	auto closest_site = 0;
	auto closest = 1.0e30f;

	// Make sure threads don't access memory outside of the image
	if((xIndex<width) && (yIndex<height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);

		#pragma unroll
		for (int index = 0; index < sites_shared->next; ++index) {

	    		 distance = sqrtf( (xIndex - sites_shared->x[index])*(xIndex - sites_shared->x[index]) +
	    						  (yIndex - sites_shared->y[index])*(yIndex - sites_shared->y[index]) );

	    		 if (distance < closest) {
	    			closest_site = index;
	    			closest = distance;
	    		}
	    }

		// Determine pixel index of closest site
		const int site_result = sites_shared->y[closest_site]*colorWidthStep + (3*sites_shared->x[closest_site]);

		// Copy site pixel value into output pixel
		output[color_tid]		= input[site_result];
		output[color_tid + 1]	= input[site_result + 1];
		output[color_tid + 2]	= input[site_result + 2];

	}
}

Voronoi::Voronoi(const cv::Mat& input) {

	const int bytes = input.step * input.rows;

	// Allocate memory on the gpu
	hipMalloc<COORDS_T>(&d_sites, sizeof(COORDS_T));
	hipMalloc<unsigned char>(&d_input, bytes);
	hipMalloc<unsigned char>(&d_output, bytes);

	// Copy host data to device
	hipMemcpy(d_input, input.ptr(), bytes, hipMemcpyHostToDevice);
}

Voronoi::~Voronoi() {
	clean();
}


void Voronoi::voronoi_gpu(cv::Mat& output){
	const int bytes = output.step * output.rows;

	// Set block size (256 threads per block)
	const dim3 block(16,16);

	const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y -1)/block.y);

	voronoi_kernel<<<grid,block>>>(d_input, d_output, output.cols, output.rows, output.step, d_sites);
	//voronoi_kernel_shared_mem<<<grid,block>>>(d_input, d_output, output.cols, output.rows, output.step, d_sites);
	hipDeviceSynchronize();

	hipMemcpy(output.ptr(), d_output, bytes, hipMemcpyDeviceToHost);

}

void Voronoi::update(COORDS_T *sites){
	// Copy new site data to the device
	hipMemcpy(d_sites, sites, sizeof(COORDS_T), hipMemcpyHostToDevice);
}

void Voronoi::clean(){
	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_sites);
}

